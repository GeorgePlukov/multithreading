#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "pa5.h"
#include "ppm.h"




int blur_radius;
int img_w;
int img_h;

char* input_ppm;
char* output_ppm;

struct Image* img_in;
struct Image* img_out;



__global__ void hello() {
	int myId = 1;
	printf("hello from %d \n",  myId);
}


int main(int argc, char** argv) {

	if (argc < 3) {
		printf("Not enough arguments\n");
		return 0;
	}


	blur_radius = atoi(argv[1]);
	input_ppm  = argv[2];
	output_ppm = argv[3];


	if (blur_radius < 1) {
		printf("blur radius too small\n");
		return 0;
	}

	init();

	printf("Image size is (width: %d, height: %d) \n", img_w, img_h);

	run();



	return 0;
}


int init() {

	//Create the two necessary Image objects
	img_in  = ImageRead(input_ppm);
	img_w = ImageWidth(img_in);
	img_h = ImageHeight(img_in);
	img_out = ImageCreate(img_w, img_h);

	return 0;
}


int run() {

	hello<<<1,1>>>();
	hipDeviceSynchronize();

	return 1;
}


int monus (int x, int y) {
	if (x - y < 0){
		return 0;
	}
	return x;
}


int maxus (int x, int y, int max) {
	if (x + y > max){
		return max;
	}
	return x + y;
}


struct Pixel averagePixels(int x, int y) {
	int minX = monus(x,blur_radius);
	int maxX = maxus(x,blur_radius,img_in->width);
	int minY = monus(y,blur_radius);
	int maxY = maxus(y,blur_radius,img_in->height);
	int red = 0;
	int green = 0;
	int blue = 0;
	int num_pixels = 0;
	int i,j;
	unsigned char r = 0, g = 0, b = 0;

	for ( i = minX; i <= maxX; i++){
		for ( j = minY; j <= maxY; j++){
			r = ImageGetPixel(img_in, i, j, 0);
			g = ImageGetPixel(img_in, i, j, 1);
			b = ImageGetPixel(img_in, i, j, 2);
			red += (int)r;
			green +=(int) g;
			blue += (int)b;
			num_pixels++;
		}
	}

	red   = floor(red / num_pixels);
	green = floor(green / num_pixels);
	blue  = floor(blue / num_pixels);

	struct Pixel jp;

	jp.x = x;
	jp.y = y;
	jp.r = red;
	jp.b = blue;
	jp.g = green;

	return jp;
}


//given a pixel, update it on the img_out
void update_image(struct Pixel pixel) {

	ImageSetPixel(img_out, pixel.x, pixel.y, 0, pixel.r);
	ImageSetPixel(img_out, pixel.x, pixel.y, 1, pixel.g);
	ImageSetPixel(img_out, pixel.x, pixel.y, 2, pixel.b);

}

void writeoutput() {
	//now that we built img_out, save it to the desired file
	ImageWrite(img_out, output_ppm);
}





void cleanup() {
	free(img_in);
	free(img_out);
}









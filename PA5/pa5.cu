#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "pa5.h"
#include "ppm.h"

__device__ int monus(int x, int y);
__device__ int maxus(int x, int y,int max);

dim3 block_dim, grid_dim;

int blur_radius;
int img_w;
int img_h;

char* input_ppm;
char* output_ppm;

struct Image* img_in;
struct Image* img_out;



__global__ void hello(int w, int h,int r, struct Pixel *in, struct Pixel *out) {
	// int myId = 1;
	int blockId =  blockIdx.y * gridDim.x + blockIdx.x;
	int threadId = blockId * blockDim.x + threadIdx.x;
	int index = 0;
	if (blockId == 1079) {
		printf("hello from blockid: %d threadId: %d \n",  blockId, threadId);
		printf("Pixel (x,y,r,g,b) :  (%d %d %d %d %d) \n",  in[blockId].x,in[blockId].y,in[blockId].r,in[blockId].g,in[blockId].b );


		int minX = monus(in[blockId].x,r);
		int maxX = maxus(in[blockId].x,r,w);
		int minY = monus(in[blockId].y,r);
		int maxY = maxus(in[blockId].y,r,h);
		int red = 0;
		int green = 0;
		int blue = 0;
		int num_pixels = 0;
		int i,j;
		unsigned char r = 0, g = 0, b = 0;

		for ( i = minX; i <= maxX; i++){
			for ( j = minY; j <= maxY; j++){

				r = in[i*w+j + threadId%3].r;
				g = in[i*w+j + threadId%3 +1].g;
				b = in[i*w+j + threadId%3 +2].b;
				red += (int)r;
				green +=(int) g;
				blue += (int)b;
				num_pixels++;
			}
		}

		red   = floor( (float) red / num_pixels );
		green = floor((float) green / num_pixels);
		blue  = floor((float) blue / num_pixels);

		struct Pixel jp;


		jp.x = in[blockId].x;
		jp.y = in[blockId].y;
		jp.r = red;
		jp.b = blue;
		jp.g = green;

		out[blockId] = jp;
	}
}


int main(int argc, char** argv) {

	if (argc < 3) {
		printf("Not enough arguments\n");
		return 0;
	}


	blur_radius = atoi(argv[1]);
	input_ppm  = argv[2];
	output_ppm = argv[3];


	if (blur_radius < 1) {
		printf("blur radius too small\n");
		return 0;
	}

	init();

	printf("Image size is (width: %d, height: %d) \n", img_w, img_h);

	run();



	return 0;
}


int init() {

	//Create the two necessary Image objects
	img_in  = ImageRead(input_ppm);
	img_w = ImageWidth(img_in);
	img_h = ImageHeight(img_in);
	img_out = ImageCreate(img_w, img_h);

	block_dim = dim3(3);
	grid_dim = dim3(img_h, img_w);

	return 0;
}


int run() {


	int num_pixels   = img_w*img_h;
	struct Pixel *pixels_host_in  = 	(Pixel *) malloc(sizeof(Pixel *)*num_pixels);
	struct Pixel *pixels_host_out = 	(Pixel *) malloc(sizeof(Pixel *)*num_pixels);

	struct Pixel *pixel_device_in ;
	struct Pixel *pixel_device_out ;


	hipMalloc((void **) &pixel_device_in,  (sizeof(Pixel *)*num_pixels));
	hipMalloc((void **) &pixel_device_out,  (sizeof(Pixel *)*num_pixels));


	// populate pixel_device_in
	int x,y;
	int index = 0;
	for (y = 0; y < img_h; y++) {
		for (x = 0; x < img_w; x++) {
			index = y*img_w+x;
			// printf("index = %d\n", index );
			pixels_host_in[index].x = x;
			pixels_host_in[index].y = y;
			pixels_host_in[index].r = ImageGetPixel(img_in, x, y, 0);
			pixels_host_in[index].g = ImageGetPixel(img_in, x, y, 1);
			pixels_host_in[index].b = ImageGetPixel(img_in, x, y, 2);

		}
	}


	hipMemcpy(pixel_device_in, pixels_host_in, sizeof(Pixel *)*num_pixels, hipMemcpyHostToDevice);

	hello<<<grid_dim, block_dim>>>(img_w, img_h, blur_radius, pixel_device_in, pixel_device_out);

	hipDeviceSynchronize();

	hipMemcpy(pixels_host_out, pixel_device_out, sizeof(Pixel *)*num_pixels, hipMemcpyDeviceToHost);



	hipFree((void*) pixel_device_in);
	hipFree((void*) pixel_device_out);

	free(pixels_host_in);
	free(pixels_host_out);

	printf("num pixels = %d\n", num_pixels );
	printf("Image size is (width: %d, height: %d) \n", img_w, img_h);


	return 1;
}


__device__ int monus (int x, int y) {
	if (x - y < 0){
		return 0;
	}
	return x;
}


__device__ int maxus (int x, int y, int max) {
	if (x + y > max){
		return max;
	}
	return x + y;
}


// __device__ struct Pixel averagePixels(int x, int y, struct Pixel *in) {
// 	int minX = monus(x,blur_radius);
// 	int maxX = maxus(x,blur_radius,img_in->width);
// 	int minY = monus(y,blur_radius);
// 	int maxY = maxus(y,blur_radius,img_in->height);
// 	int red = 0;
// 	int green = 0;
// 	int blue = 0;
// 	int num_pixels = 0;
// 	int i,j;
// 	unsigned char r = 0, g = 0, b = 0;
//
// 	for ( i = minX; i <= maxX; i++){
// 		for ( j = minY; j <= maxY; j++){
//
//
// 			r = ImageGetPixel(img_in, i, j, 0);
// 			g = ImageGetPixel(img_in, i, j, 1);
// 			b = ImageGetPixel(img_in, i, j, 2);
// 			red += (int)r;
// 			green +=(int) g;
// 			blue += (int)b;
// 			num_pixels++;
// 		}
// 	}
//
// 	red   = floor(red / num_pixels);
// 	green = floor(green / num_pixels);
// 	blue  = floor(blue / num_pixels);
//
// 	struct Pixel jp;
//
// 	jp.x = x;
// 	jp.y = y;
// 	jp.r = red;
// 	jp.b = blue;
// 	jp.g = green;
//
// 	return jp;
// }


//given a pixel, update it on the img_out
void update_image(struct Pixel pixel) {

	ImageSetPixel(img_out, pixel.x, pixel.y, 0, pixel.r);
	ImageSetPixel(img_out, pixel.x, pixel.y, 1, pixel.g);
	ImageSetPixel(img_out, pixel.x, pixel.y, 2, pixel.b);

}

void writeoutput() {
	//now that we built img_out, save it to the desired file
	ImageWrite(img_out, output_ppm);
}





void cleanup() {
	free(img_in);
	free(img_out);
}

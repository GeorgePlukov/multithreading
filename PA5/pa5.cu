#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "pa5.h"
#include "ppm.h"

__device__ int monus(int x, int y);
__device__ int maxus(int x, int y,int max);

dim3 block_dim, grid_dim;

int blur_radius;
int img_w;
int img_h;

char* input_ppm;
char* output_ppm;

struct Image* img_in;
struct Image* img_out;


/*
	GPU function for blurring an image of width w and height r
	Blur radius given by r
	in: Pixel values to be used for calculations
	out: Blurred pixel values
*/
__global__ void blur_img(int w, int h,int r, struct Pixel *in, struct Pixel *out) {
	int blockId =  blockIdx.y * gridDim.x + blockIdx.x;
	int threadId = blockId * blockDim.x + threadIdx.x;

	if (blockId < h*w) {
		int minX = monus(in[blockId].x,r);
		int maxX = maxus(in[blockId].x,r,w);
		int minY = monus(in[blockId].y,r);
		int maxY = maxus(in[blockId].y,r,h);
		int red = 0;
		int green = 0;
		int blue = 0;
		int num_pixels = 0;
		int x,y;
		int index = 0;
		unsigned char r = 0, g = 0, b = 0;

		for ( y = minY; y <= maxY; y++ ){
			for (x = minX; x <= maxX; x++){
				index = y*w+x;
				r = in[index + threadId%3].r;
				g = in[index + threadId%3 +1].g;
				b = in[index + threadId%3 +2].b;
				red += (int)r;
				green +=(int) g;
				blue += (int)b;
				num_pixels++;
			}
		}

		red   = floor( (float) red / num_pixels );
		green = floor( (float) green / num_pixels);
		blue  = floor( (float) blue / num_pixels);

		struct Pixel jp;


		jp.x = in[blockId].x;
		jp.y = in[blockId].y;
		jp.r = red;
		jp.b = blue;
		jp.g = green;

		out[blockId] = jp;
	}
}


int main(int argc, char** argv) {

	if (argc < 3) {
		printf("Not enough arguments\n");
		return 0;
	}

	blur_radius = atoi(argv[1]);
	input_ppm  = argv[2];
	output_ppm = argv[3];

	if (blur_radius < 1) {
		printf("blur radius too small\n");
		return 0;
	}

	init();

	run();


	return 0;
}


int init() {

	//Create the two necessary Image objects
	img_in  = ImageRead(input_ppm);
	img_w 	= ImageWidth(img_in);
	img_h 	= ImageHeight(img_in);
	img_out = ImageCreate(img_w, img_h);

	//Setup the grid and block sizes based on image width and height
	block_dim = dim3(3);
	grid_dim = dim3(img_h, img_w);

	return 0;
}




/*
	Setup two copies of pixels array to work on the host and device.
	Call GPU function to blur all the images
	Write the result
*/
int run() {


	int num_pixels = img_w*img_h;

	struct Pixel *pixels_host_in  = (Pixel *) malloc(sizeof(Pixel)*num_pixels);
	struct Pixel *pixels_host_out = (Pixel *) malloc(sizeof(Pixel)*num_pixels);

	struct Pixel *pixel_device_in ;
	struct Pixel *pixel_device_out ;


	hipMalloc((void **) &pixel_device_in,  (sizeof(Pixel)*num_pixels));
	hipMalloc((void **) &pixel_device_out, (sizeof(Pixel)*num_pixels));


	// populate pixel_device_in
	int x,y;
	int index = 0;
	for (y = 0; y < img_h; y++) {
		for (x = 0; x < img_w; x++) {
			index = y*img_w + x;
			pixels_host_in[index].x = x;
			pixels_host_in[index].y = y;
			pixels_host_in[index].r = ImageGetPixel(img_in, x, y, 0);
			pixels_host_in[index].g = ImageGetPixel(img_in, x, y, 1);
			pixels_host_in[index].b = ImageGetPixel(img_in, x, y, 2);

		}
	}


	hipMemcpy(pixel_device_in, (Pixel*)pixels_host_in, sizeof(Pixel)*num_pixels, hipMemcpyHostToDevice);

	blur_img<<<grid_dim, block_dim>>>(img_w, 
									  img_h, 
									  blur_radius, 
									  pixel_device_in, 
									  pixel_device_out);

	hipDeviceSynchronize();
	hipGetLastError();

	hipMemcpy((Pixel*)pixels_host_out, pixel_device_out, sizeof(Pixel)*num_pixels, hipMemcpyDeviceToHost);



	//write result back to ppm img
	for (y = 0; y < img_h; y++) {
		for (x = 0; x < img_w; x++) {
			index = y*img_w+x;
			update_image(pixels_host_out[index]);
		}	
	}
	
	

	ImageWrite(img_out, output_ppm);

	hipFree((void*) pixel_device_in);
	hipFree((void*) pixel_device_out);

	free(pixels_host_in);
	free(pixels_host_out);
	free(img_in);
	free(img_out);

	printf("Image size is (width: %d, height: %d) \n", img_w, img_h);
	printf("Number of pixels blurred: %d\n", num_pixels );
	printf("Blurred image written to %s\n", output_ppm);

	return 1;
}


__device__ int monus (int x, int y) {
	if (x - y < 0){
		return 0;
	}
	return x;
}


__device__ int maxus (int x, int y, int max) {
	if (x + y > max){
		return max;
	}
	return x + y;
}




//given a pixel, update it on the img_out
void update_image(struct Pixel pixel) {
	ImageSetPixel(img_out, pixel.x, pixel.y, 0, pixel.r);
	ImageSetPixel(img_out, pixel.x, pixel.y, 1, pixel.g);
	ImageSetPixel(img_out, pixel.x, pixel.y, 2, pixel.b);
}




